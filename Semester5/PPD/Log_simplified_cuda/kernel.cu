
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <fstream>

using namespace std;


__global__ void applyFilterGPU(int* in, int* out, int lines, int cols, int channels) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < lines*cols*channels) {
		//out[id] = in[id]+20;

		//apply kernel
		int line = id / (cols*channels);
		int col = (id % (cols*channels)) / channels;
		int channel = id % channels;
		
			
		int value = 0;
		if (line > 1 && line < (lines-2) && col > 1 && col < (cols-2)) {
			//center
			value += 16*in[id];
			
			//north
			value += -2*in[id - cols*channels];

			//east
			value += -2*in[id+channels];

			//south
			value += -2*in[id+cols*channels];

			//west
			value += -2*in[id-channels];
			
			//north west
			value += -1*in[id - cols*channels - channels];

			//north east
			value += -1*in[id - cols*channels + channels];

			//south east
			value += -1*in[id + cols*channels + channels];

			//south west
			value += -1*in[id + cols*channels - channels];
			
			//3rd level convolutions
			//top

			value += 0 *in[id - 2* cols * channels - 2*channels];
			value += 0 * in[id - 2 * cols * channels -  channels];
			value += -1 * in[id - 2 * cols * channels];
			value += 0 * in[id - 2 * cols * channels + channels];
			value += 0 * in[id - 2 * cols * channels + 2 * channels];

			//right
			value += 0* in[id - 1 * cols * channels + 2 * channels];
			value += -1 * in[id - 0 * cols * channels + 2 * channels];
			value += 0 * in[id + 1 * cols * channels + 2 * channels];

			//left
			value += 0 * in[id - 1 * cols * channels - 2 * channels];
			value += -1 * in[id - 0 * cols * channels - 2 * channels];
			value += 0 * in[id + 1 * cols * channels - 2 * channels];


			//down
			value += 0 * in[id + 2 * cols * channels - 2 * channels];
			value += 0 * in[id + 2 * cols * channels - channels];
			value += -1 * in[id + 2 * cols * channels];
			value += 0 * in[id + 2 * cols * channels + channels];
			value += 0 * in[id + 2 * cols * channels + 2 * channels];

			value = value / 1;
		}
		else {
			value = 0;
		}

		out[id] = value;
	}
}



int* flatten(int*** matrix, int lines, int cols, int channels) {

	int* flat = (int*)malloc(lines*cols*channels * sizeof(int));
	int id = 0;
	for (int i = 0; i < lines; i++) {
		for (int j = 0; j < cols; j++) {
			for (int c = 0; c < channels; c++) {
				flat[id] = matrix[i][j][c];
				id++;
			}
		}
	}
	return flat;
}

int*** unflatten(int* arr, int lines, int cols, int channels) {
	int*** img = (int***)malloc(lines * sizeof(int**));
	int id = 0;

	for (int i = 0; i < lines; i++) {
		img[i] = (int**)malloc(cols * sizeof(int*));

		for (int j = 0; j < cols; j++) {
			img[i][j] = (int*)malloc(channels * sizeof(int));

			for (int c = 0; c < channels; c++) {
				img[i][j][c] = arr[id];
				id++;
			}
		}
	}
	return img;
}

void applyFilter() {

	//read the matrix
	ifstream in("pixels.txt");
	int lines, cols, channels;
	in >> lines >> cols >> channels;

	int BLOCK_SIZE = 1000;

	int blockCount = ((lines*cols*channels) / BLOCK_SIZE) + 1;
	

	//matrix read
	int*** matrix = (int***)malloc(lines * sizeof(int**));
	for (int i = 0; i < lines; i++) {
		matrix[i] = (int**)malloc(cols * sizeof(int*));

		for (int j = 0; j < cols; j++) {
			int* line = (int*)malloc(channels * sizeof(int));

			in >> line[0] >> line[1] >> line[2];
			
			matrix[i][j] = line;
		}
	}

	int size = lines * cols*channels;
	//flatten matrix
	int* flatMatrix = flatten(matrix, lines, cols, channels);
	int* result = (int*)malloc(size * sizeof(int));

	//copy flatMatrix to the device array
	int* deviceMatrix, *deviceResult;
	hipMalloc(&deviceMatrix, size * sizeof(int));
	hipMalloc(&deviceResult, size * sizeof(int));

	hipMemcpy(
		deviceMatrix, flatMatrix,
		size * sizeof(int),
		hipMemcpyHostToDevice
	);

	//call the filter function
	applyFilterGPU <<< blockCount,1000 >>> (deviceMatrix, deviceResult, lines, cols, channels);

	//copy from deviceResult in result
	hipMemcpy(
		result, deviceResult,
		size * sizeof(int),
		hipMemcpyDeviceToHost
	);

	int*** img = unflatten(result, lines, cols, channels);





	ofstream out("pixels.txt");
	out << lines << " " << cols << " " << channels << "\n";
	for (int i = 0; i < lines; i++) {
		for (int j = 0; j < cols; j++) {
			for (int k = 0; k < channels; k++) {
				out << img[i][j][k] << " ";
			}
			out << "\n";
		}
	}

	out.close();
}
int main()
{
	char* inImgPath = "/home/geo/Programming/University/Semester5/PPD/Log_simplified_cuda/dia.jpg";
	char* outImgPath = "/home/geo/Programming/University/Semester5/PPD/Log_simplified_cuda/dia.jpg";
	char* inCommand = "conda activate ai & python in.py /home/geo/Programming/University/Semester5/PPD/Log_simplified_cuda/dia.jpg" ;
	char* outCommand = "conda activate ai & python out.py /home/geo/Programming/University/Semester5/PPD/Log_simplified_cuda/dia1.jpg" ;
	
	//system(inCommand);			//read pixel values to pixels.txt

	applyFilter();
	
	//system(outCommand);			//write pixel values to image



	return 0;
}
